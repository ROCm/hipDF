#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2022-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
// MIT License
//
// Modifications Copyright (C) 2023-2024 Advanced Micro Devices, Inc. All rights reserved.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include "rolling.cuh"

#include <cudf/detail/aggregation/aggregation.hpp>
#include <cudf/utilities/default_stream.hpp>

#include <cudf_test/column_utilities.hpp>

#include <thrust/extrema.h>

namespace cudf::detail {

// Applies a fixed-size rolling window function to the values in a column.
std::unique_ptr<column> rolling_window(column_view const& input,
                                       column_view const& default_outputs,
                                       size_type preceding_window,
                                       size_type following_window,
                                       size_type min_periods,
                                       rolling_aggregation const& agg,
                                       rmm::cuda_stream_view stream,
                                       rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();

  if (input.is_empty()) { return cudf::detail::empty_output_for_rolling_aggregation(input, agg); }

  CUDF_EXPECTS((min_periods >= 0), "min_periods must be non-negative");

  CUDF_EXPECTS((default_outputs.is_empty() || default_outputs.size() == input.size()),
               "Defaults column must be either empty or have as many rows as the input column.");

  CUDF_EXPECTS(-(preceding_window - 1) <= following_window,
               "Preceding window bounds must precede the following window bounds.");

  if (agg.kind == aggregation::CUDA || agg.kind == aggregation::PTX) {
    // TODO: In future, might need to clamp preceding/following to column boundaries.
    return cudf::detail::rolling_window_udf(input,
                                            preceding_window,
                                            "cudf::size_type",
                                            following_window,
                                            "cudf::size_type",
                                            min_periods,
                                            agg,
                                            stream,
                                            mr);
  } else {
    // Clamp preceding/following to column boundaries.
    // E.g. If preceding_window == 2, then for a column of 5 elements, preceding_window will be:
    //      [1, 2, 2, 2, 1]

    auto const preceding_calc = [preceding_window] __device__(size_type i) {
      return thrust::min(i + 1, preceding_window);
    };

    auto const following_calc = [col_size = input.size(),
                                 following_window] __device__(size_type i) {
      return thrust::min(col_size - i - 1, following_window);
    };

    auto const preceding_column = expand_to_column(preceding_calc, input.size(), stream);
    auto const following_column = expand_to_column(following_calc, input.size(), stream);
    return cudf::detail::rolling_window(input,
                                        default_outputs,
                                        preceding_column->view().begin<cudf::size_type>(),
                                        following_column->view().begin<cudf::size_type>(),
                                        min_periods,
                                        agg,
                                        stream,
                                        mr);
  }
}
}  // namespace cudf::detail
