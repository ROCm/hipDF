#include "hip/hip_runtime.h"
#include "cudf/hip/hip_runtime.h"
/*
 * Copyright (c) 2018-2023, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
// MIT License
//
// Modifications Copyright (C) 2023-2024 Advanced Micro Devices, Inc. All rights reserved.
//
// Permission is hereby granted, free of charge, to any person obtaining a copy
// of this software and associated documentation files (the "Software"), to deal
// in the Software without restriction, including without limitation the rights
// to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
// copies of the Software, and to permit persons to whom the Software is
// furnished to do so, subject to the following conditions:
//
// The above copyright notice and this permission notice shall be included in all
// copies or substantial portions of the Software.
//
// THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
// IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
// FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
// AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
// LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
// OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
// SOFTWARE.

#include <cudf_test/base_fixture.hpp>
#include <cudf_test/default_stream.hpp>
#include <cudf_test/stream_checking_resource_adaptor.hpp>

#include <cudf/filling.hpp>
#include <cudf/utilities/default_stream.hpp>
#include <cudf/utilities/error.hpp>

#include <rmm/hip_stream.hpp>

TEST(ExpectsTest, FalseCondition)
{
  EXPECT_THROW(CUDF_EXPECTS(false, "condition is false"), cudf::logic_error);
}

TEST(ExpectsTest, TrueCondition) { EXPECT_NO_THROW(CUDF_EXPECTS(true, "condition is true")); }

TEST(CudaTryTest, Error) { EXPECT_THROW(CUDF_CUDA_TRY(hipErrorLaunchFailure), cudf::cuda_error); }

TEST(CudaTryTest, Success) { EXPECT_NO_THROW(CUDF_CUDA_TRY(hipSuccess)); }

TEST(StreamCheck, success) { EXPECT_NO_THROW(CUDF_CHECK_CUDA(0)); }

namespace {
// Some silly kernel that will cause an error
void __global__ test_kernel(int* data) { data[threadIdx.x] = threadIdx.x; }
}  // namespace

// In a release build and without explicit synchronization, CUDF_CHECK_CUDA may
// or may not fail on erroneous asynchronous CUDA calls. Invoke
// hipStreamSynchronize to guarantee failure on error. In a non-release build,
// CUDF_CHECK_CUDA deterministically fails on erroneous asynchronous CUDA
// calls.
TEST(StreamCheck, FailedKernel)
{

  if constexpr(cudf::HIP_PLATFORM_AMD) {
    GTEST_SKIP() << "This test is presently not supported on AMD platform (internal issue 9)"; 
  }

  rmm::hip_stream stream;
  int a;
  // TODO(HIP/AMD): test_kernel<<<0, 0, 0, stream.value()>>>(&a);
  test_kernel<<<1, 1, 0, stream.value()>>>(&a);
#ifdef NDEBUG
  stream.synchronize();
#endif
  EXPECT_THROW(CUDF_CHECK_CUDA(stream.value()), cudf::cuda_error);
}

TEST(StreamCheck, CatchFailedKernel)
{
  rmm::hip_stream stream;
  int a;
  test_kernel<<<0, 0, 0, stream.value()>>>(&a);
#ifndef NDEBUG
  stream.synchronize();
#endif
  EXPECT_THROW(CUDF_CHECK_CUDA(stream.value()), cudf::cuda_error);
}

// TODO(HIP/AMD): should we use s_trap here?
__global__ void kernel() { abort(); }

TEST(DeathTest, CudaFatalError)
{
  testing::FLAGS_gtest_death_test_style = "threadsafe";
  auto call_kernel                      = []() {
    kernel<<<1, 1, 0, cudf::get_default_stream().value()>>>();
    try {
      CUDF_CUDA_TRY(hipDeviceSynchronize());
    } catch (const cudf::fatal_cuda_error& fe) {
      std::abort();
    }
  };
  ASSERT_DEATH(call_kernel(), "");
}

#ifndef NDEBUG

__global__ void assert_false_kernel() { cudf_assert(false && "this kernel should die"); }

__global__ void assert_true_kernel() { cudf_assert(true && "this kernel should live"); }

TEST(DebugAssertDeathTest, cudf_assert_false)
{
  testing::FLAGS_gtest_death_test_style = "threadsafe";

  auto call_kernel = []() {
    assert_false_kernel<<<1, 1>>>();

    // Kernel should fail with `hipErrorAssert`
    // This error invalidates the current device context, so we need to kill
    // the current process. Running with EXPECT_DEATH spawns a new process for
    // each attempted kernel launch
    if (hipErrorAssert == hipDeviceSynchronize()) { std::abort(); }

    // If we reach this point, the cudf_assert didn't work so we exit normally, which will cause
    // EXPECT_DEATH to fail.
  };

  EXPECT_DEATH(call_kernel(), "this kernel should die");
}

TEST(DebugAssert, cudf_assert_true)
{
  assert_true_kernel<<<1, 1>>>();
  ASSERT_EQ(hipSuccess, hipDeviceSynchronize());
}

#endif

// These tests don't use CUDF_TEST_PROGRAM_MAIN because :
// 1.) They don't need the RMM Pool
// 2.) The RMM Pool interferes with the death test
int main(int argc, char** argv)
{
  ::testing::InitGoogleTest(&argc, argv);
  auto const cmd_opts    = parse_cudf_test_opts(argc, argv);
  auto const stream_mode = cmd_opts["stream_mode"].as<std::string>();
  if ((stream_mode == "new_cudf_default") || (stream_mode == "new_testing_default")) {
    auto resource                      = rmm::mr::get_current_device_resource();
    auto const stream_error_mode       = cmd_opts["stream_error_mode"].as<std::string>();
    auto const error_on_invalid_stream = (stream_error_mode == "error");
    auto const check_default_stream    = (stream_mode == "new_cudf_default");
    auto adaptor                       = make_stream_checking_resource_adaptor(
      resource, error_on_invalid_stream, check_default_stream);
    rmm::mr::set_current_device_resource(&adaptor);
  }
  return RUN_ALL_TESTS();
}
